#include <bits/stdc++.h>
#include <stdio.h>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include "hip/hip_runtime.h"


#define NUM_TEST_FILES 1
#define GAP_SCORE -1
#define BLOCK_X_Y_DIM 32

__constant__ signed char c_s[16];

__device__ signed char base_to_val(char B) {
  // Assume 'A' unless proven otherwise.
  signed char ret = 0;
  if (B == 'G')
    ret = 1;
  if (B == 'C')
    ret = 2;
  if (B == 'T')
    ret = 3;
  return ret;
}

__device__ signed char nw_get_sim(char Ai, char Bi) {
  return c_s[base_to_val(Ai) * 4 + base_to_val(Bi)];
}

// Call this kernel "qlen + tlen - 1" times, then matrix will be done.
__global__ void nw_shotgun_scoring_kernel(
  char * t,
  char * q,
  uint32_t tlen,
  uint32_t qlen,
  signed char mis_or_ind,
  int * score_mat
) {
  // Get global and local thread index.
  int32_t g_tx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int32_t g_ty = (blockIdx.y * blockDim.y) + threadIdx.y;
  int32_t l_tx = threadIdx.x;
  int32_t l_ty = threadIdx.y;

  // Matrix dims.
  int32_t mat_w = tlen + 1;

  // Shared memory.
  __shared__ int s_score_mat[32][32];
  __shared__ char s_t[16];
  __shared__ char s_q[16];

  // Fill score matrix shared memory.
  if (g_tx == 0 && g_ty <= qlen)
    s_score_mat[l_ty][l_tx] = g_ty * mis_or_ind;
  if (g_ty == 0 && g_tx <= tlen)
    s_score_mat[l_ty][l_tx] = g_tx * mis_or_ind;
  if (g_tx > 0 && g_tx <= tlen && g_ty > 0 && g_ty <= qlen)
    s_score_mat[l_ty][l_tx] = score_mat[mat_w * g_ty + g_tx];

  // Fill target/query shared memory.
  if (g_tx == 0 && g_ty > 0 && g_ty <= qlen)
    s_q[l_ty - 1] = q[g_ty - 1];
  if (g_ty == 0 && g_tx > 0 && g_tx <= tlen)
    s_t[l_tx - 1] = t[g_tx - 1];

  // Ensure shared memory is filled before going on.
  __syncthreads();

  // If we are not a border thread then shotgun
  // compute the matrix, be it correct or not.
  if (g_tx > 0 && g_tx <= tlen && g_ty > 0 && g_ty <= qlen) {
    int match = s_score_mat[l_ty - 1][l_tx - 1] + nw_get_sim(s_q[l_ty - 1], s_t[l_tx - 1]);
    int del = s_score_mat[l_ty - 1][l_tx] + mis_or_ind;
    int ins = s_score_mat[l_ty][l_tx - 1] + mis_or_ind;
    int cell = match > del ? match : del;
    cell = cell > ins ? cell : ins;
    s_score_mat[l_ty][l_tx] = cell;
  }

  // Coalesced writeback.
  __syncthreads();
  if (g_tx <= tlen && g_ty <= qlen)
    score_mat[mat_w * g_ty + g_tx] = s_score_mat[l_ty][l_tx];
}

void nw_gpu_man(
  char * t,
  char * q,
  uint32_t tlen,
  uint32_t qlen,
  signed char mis_or_ind
) {
  // Device memory pointers.
  char * t_d;
  char * q_d;
  int * score_mat_d;

  // Malloc space on GPU.
  hipMalloc((void **) & t_d, tlen * sizeof(char));
  hipMalloc((void **) & q_d, qlen * sizeof(char));
  hipMalloc((void **) & score_mat_d, (qlen + 1) * (tlen + 1) * sizeof(int));

  // Copy to GPU.
  hipMemcpy(t_d, t, tlen * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(q_d, q, qlen * sizeof(char), hipMemcpyHostToDevice);

  // Launch compute kernel.
  dim3 GridDim(ceil((tlen + 1) / ((float) 32)), ceil((qlen + 1) / ((float) 32)));
  dim3 BlockDim(32, 32);
  for (uint32_t i = 0; i < qlen + tlen - 1; ++i) {
    nw_shotgun_scoring_kernel <<<GridDim, BlockDim>>>
      (t_d, q_d, tlen, qlen, mis_or_ind, score_mat_d);
  }

  // // TEMP: UNCOMMENT FOR MATRIX PRINTING!
  // int * score_mat = new int [(qlen + 1) * (tlen + 1)];
  // cudaMemcpy(score_mat, score_mat_d, (qlen + 1) * (tlen + 1) * sizeof(int), cudaMemcpyDeviceToHost);
  // for (int i = 0; i <= qlen; ++i) {
  //   for (int j = 0; j <= tlen; ++j)
  //     std::cout << std::setfill(' ') << std::setw(5)
  //       << score_mat[(tlen + 1) * i + j] << " ";
  //   std::cout << std::endl;
  // }

  // Clean up.
  hipFree(t_d);
  hipFree(q_d);
  hipFree(score_mat_d);
}

int main() {
  std::string input_line;
  uint32_t tlen = 0;
  uint32_t qlen = 0;
  char * t = NULL;
  char * q = NULL;
  signed char * s = NULL;
  // Read in similarity matrix file.
  std::string sim_file = "datasets/similarity.txt";
  std::ifstream sim_file_stream(sim_file);
  s = new signed char[16];
  unsigned char sim_cnt = 0;
  while (std::getline(sim_file_stream, input_line)) {
    s[sim_cnt] = std::stoi(input_line);
    ++sim_cnt;
  }
  // Write similarity matrix to constant CUDA memory.
  hipMemcpyToSymbol(HIP_SYMBOL(c_s), s, 16 * sizeof(signed char));
  // Run through test file.
  for (uint32_t i = 0; i < NUM_TEST_FILES; ++i) {
    std::string test_file = "datasets/" + std::to_string(i) + ".txt";
    std::ifstream test_file_stream(test_file);
    uint32_t test_cnt = 0;
    while (std::getline(test_file_stream, input_line)) {
      if (test_cnt == 0) {
        tlen = std::stoll(input_line);
        t = new char [tlen + 1];
      }
      if (test_cnt == 1) {
        qlen = std::stoll(input_line);
        q = new char [qlen + 1];
      }
      if (test_cnt == 2)
        strcpy(t, input_line.c_str());
      if (test_cnt == 3)
        strcpy(q, input_line.c_str());
      ++test_cnt;
    }
    nw_gpu_man(t, q, tlen, qlen, GAP_SCORE);
    delete [] q;
    delete [] t;
  }
  delete [] s;
  return 0;
}